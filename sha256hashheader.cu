
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <chrono>



// sha256 constants
const unsigned int sha256_k[64] = //UL = uint32
            {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5,
             0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
             0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3,
             0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
             0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc,
             0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
             0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
             0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
             0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13,
             0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
             0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3,
             0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
             0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5,
             0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
             0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208,
             0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

const unsigned int sha256_h[8] =
            {0x6a09e667, 0xbb67ae85, 0x3c6ef372, 0xa54ff53a, 0x510e527f, 0x9b05688c, 0x1f83d9ab, 0x5be0cd19};



__global__
void hash_header(const unsigned int* sha_k, const unsigned int* sha_h, const unsigned int* target, const unsigned int* header, unsigned int* nonce) {

    // Get the index of the current element to be processed
    // int gid = get_global_id(0);
    // unsigned int threadsPerBlock  = blockDim.x * blockDim.y;
    // unsigned int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y; // (alternatively: threadIdx.y + blockDim.y * threadIdx.x);
    // unsigned int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y; //  (alternatively: blockIdx.y  + gridDim.y  * blockIdx.x);
    // unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
    // unique block index inside a 3D block grid
    const unsigned long long int blockId = blockIdx.x //1D
            + blockIdx.y * gridDim.x //2D
            + gridDim.x * gridDim.y * blockIdx.z; //3D

    // global unique thread index, block dimension uses only x-coordinate
    const unsigned long long int gid = blockId * blockDim.x + threadIdx.x;

    unsigned int testNonce = 0xFFFFFFFF - gid;
    testNonce = 4189752839;


    int length = 80;
    int i = 0;
    unsigned int schedules_orig[2][64];

//     compression variables
    unsigned int sha256_h_buffer_orig[8] = {sha_h[0],sha_h[1],sha_h[2],sha_h[3],sha_h[4],sha_h[5],sha_h[6],sha_h[7]};
    unsigned int* sha256_h_buffer = sha256_h_buffer_orig;

    unsigned int buf[8];

    // ================================================================================================================================
    // ================================================================================================================================
    // sha part 1: message one

    unsigned int* schedules = schedules_orig[0];
    for(i = 0; i < 16; i++){
        schedules[i] = header[i];
    }

    // compression
    // fill the schedule arrays
    for(i = 16; i < 64; i++) {
        unsigned int* s0v = &schedules[i-15];
        unsigned int* s1v = &schedules[i-2];
        unsigned int s0 = ((*s0v >> 7)|(*s0v << 25)) ^ ((*s0v >> 18)|(*s0v << 14)) ^ (*s0v >> 3);
        unsigned int s1 = ((*s1v >> 17)|(*s1v << 15)) ^ ((*s1v >> 19)|(*s1v << 13)) ^ (*s1v >> 10);
        schedules[i] = schedules[i-16] + s0 + schedules[i-7] + s1;
//                schedules[i] = schedules[i-16] + schedules[i-7] + SHA256_F4(schedules[i-2]) + SHA256_F3(schedules[i-15]);
    }

    for(i = 0; i < 8; i++) {
        buf[i] = sha256_h_buffer[i];
    }

    // COMPRESSION
    for(i = 0; i < 64; i++) {
        unsigned int f1 = ((buf[0] >> 2)|(buf[0] << 30)) ^ ((buf[0] >> 13)|(buf[0] << 19)) ^ ((buf[0] >> 22)|(buf[0] << 10));
        unsigned int maj = (buf[0] & buf[1]) ^ (buf[0] & buf[2]) ^ (buf[1] & buf[2]);
        unsigned int temp2 = f1 + maj;

        unsigned int f2 = ((buf[4] >> 6)|(buf[4] << 26)) ^ ((buf[4] >> 11)|(buf[4] << 21)) ^ ((buf[4] >> 25)|(buf[4] << 7));
        unsigned int ch = ((buf[4] & buf[5]) ^ (~buf[4] & buf[6]));
        unsigned int temp1 = buf[7] + f2 + ch + sha_k[i] + schedules[i];

//            unsigned int temp2 = SHA256_F1(buf[0]) + SHA2_MAJ(buf[0], buf[1], buf[2]);
//            unsigned int temp1 = buf[7] + SHA256_F2(buf[4]) + SHA2_CH(buf[4],buf[5],buf[6]) + sha256_k[i] + schedules2[i];
        buf[7] = buf[6];
        buf[6] = buf[5];
        buf[5] = buf[4];
        buf[4] = buf[3]+temp1;
        buf[3] = buf[2];
        buf[2] = buf[1];
        buf[1] = buf[0];
        buf[0] = temp1+temp2;
    }

    for(i = 0; i < 8; i++) {
//        sha256_h_buffer[i] += buf[i];
        buf[i] += sha256_h_buffer[i];
    }

    // ================================================================================================================================
    // ================================================================================================================================
    // sha part 1: message two

    unsigned int* schedulesp2 = schedules_orig[1];
    // blank all fields, only part blank needed
    for(i = 3; i < 16; i++) schedulesp2[i] = 0;

    for(i = 16; i < 19; i++){
        schedulesp2[i] = header[i];
        // pack into 8 byte/32 bits array
        // schedules[i] = (*(header + (i << 2) + (bl << 6)) << 24) + (*(header + (i << 2) + 1 + (bl << 6)) << 16) + (*(header + (i << 2) + 2 + (bl << 6)) << 8) + (*(header + (i << 2) + 3 + (bl << 6)));
    }

    schedulesp2[4] = 0x80000000;
    schedulesp2[15] = length*8;

    // change nonce here
    unsigned int testNonceBigEndian = (testNonce << 24) + (((testNonce >> 8) << 24) >> 8) + (((testNonce >> 16) << 24) >> 16) + (testNonce >> 24);
    schedulesp2[3] = testNonceBigEndian;


    // compression
    // fill the schedule arrays
    for(i = 16; i < 64; i++) {
        unsigned int* s0v = &schedulesp2[i-15];
        unsigned int* s1v = &schedulesp2[i-2];
        unsigned int s0 = ((*s0v >> 7)|(*s0v << 25)) ^ ((*s0v >> 18)|(*s0v << 14)) ^ (*s0v >> 3);
        unsigned int s1 = ((*s1v >> 17)|(*s1v << 15)) ^ ((*s1v >> 19)|(*s1v << 13)) ^ (*s1v >> 10);
        schedulesp2[i] = schedulesp2[i-16] + s0 + schedulesp2[i-7] + s1;
//                schedules[i] = schedules[i-16] + schedules[i-7] + SHA256_F4(schedules[i-2]) + SHA256_F3(schedules[i-15]);
    }

//    for(i = 0; i < 8; i++) {
//        buf[i] = sha256_h_buffer[i];
//    }

    // COMPRESSION
    for(i = 0; i < 64; i++) {
        unsigned int f1 = ((buf[0] >> 2)|(buf[0] << 30)) ^ ((buf[0] >> 13)|(buf[0] << 19)) ^ ((buf[0] >> 22)|(buf[0] << 10));
        unsigned int maj = (buf[0] & buf[1]) ^ (buf[0] & buf[2]) ^ (buf[1] & buf[2]);
        unsigned int temp2 = f1 + maj;

        unsigned int f2 = ((buf[4] >> 6)|(buf[4] << 26)) ^ ((buf[4] >> 11)|(buf[4] << 21)) ^ ((buf[4] >> 25)|(buf[4] << 7));
        unsigned int ch = ((buf[4] & buf[5]) ^ (~buf[4] & buf[6]));
        unsigned int temp1 = buf[7] + f2 + ch + sha_k[i] + schedulesp2[i];

//            unsigned int temp2 = SHA256_F1(buf[0]) + SHA2_MAJ(buf[0], buf[1], buf[2]);
//            unsigned int temp1 = buf[7] + SHA256_F2(buf[4]) + SHA2_CH(buf[4],buf[5],buf[6]) + sha256_k[i] + schedules2[i];
        buf[7] = buf[6];
        buf[6] = buf[5];
        buf[5] = buf[4];
        buf[4] = buf[3]+temp1;
        buf[3] = buf[2];
        buf[2] = buf[1];
        buf[1] = buf[0];
        buf[0] = temp1+temp2;
    }

    for(i = 0; i < 8; i++) {
        sha256_h_buffer[i] += buf[i];
    }

    // ================================================================================================================================
    // ================================================================================================================================
    // sha part 2: hashing the hash

    // SECOND RUNN
    // compression variables
    unsigned int sha256_h_buffer_orig2[8] = {sha_h[0],sha_h[1],sha_h[2],sha_h[3],sha_h[4],sha_h[5],sha_h[6],sha_h[7]};
    unsigned int* sha256_h_buffer2 = sha256_h_buffer_orig2;

    unsigned int schedules2[64];
    // blank all fields, only partial needed
    for(i = 8; i < 16; i++) schedules2[i] = 0;

//        memcpy(&schedules2, &sha256_h_buffer_orig, 32);
    for (i = 0; i < 8; i++) {
        schedules2[i] = sha256_h_buffer_orig[i];
    }
//    schedules2[0] = sha256_h_buffer_orig[0];
//    schedules2[1] = sha256_h_buffer_orig[1];
//    schedules2[2] = sha256_h_buffer_orig[2];
//    schedules2[3] = sha256_h_buffer_orig[3];
//    schedules2[4] = sha256_h_buffer_orig[4];
//    schedules2[5] = sha256_h_buffer_orig[5];
//    schedules2[6] = sha256_h_buffer_orig[6];
//    schedules2[7] = sha256_h_buffer_orig[7];

    // add a single bit to end of values
    schedules2[8] = 0x80000000;
    schedules2[15] = 256;


    // compression
    // fill the schedule arrays
    for(i = 16; i < 64; i++) {
            unsigned int* s0v = &schedules2[i-15];
            unsigned int* s1v = &schedules2[i-2];
            unsigned int s0 = ((*s0v >> 7)|(*s0v << 25)) ^ ((*s0v >> 18)|(*s0v << 14)) ^ (*s0v >> 3);
            unsigned int s1 = ((*s1v >> 17)|(*s1v << 15)) ^ ((*s1v >> 19)|(*s1v << 13)) ^ (*s1v >> 10);
            schedules2[i] = schedules2[i-16] + s0 + schedules2[i-7] + s1;
//            schedules2[i] = schedules2[i-16] + schedules2[i-7] + SHA256_F4(schedules2[i-2]) + SHA256_F3(schedules2[i-15]);
    }

    unsigned int buf2[8];
    for(i = 0; i < 8; i++) {
        buf2[i] = sha256_h_buffer2[i];
    }

    // COMPRESSION
    for(i = 0; i < 64; i++) {
        unsigned int f1 = ((buf2[0] >> 2)|(buf2[0] << 30)) ^ ((buf2[0] >> 13)|(buf2[0] << 19)) ^ ((buf2[0] >> 22)|(buf2[0] << 10));
        unsigned int maj = (buf2[0] & buf2[1]) ^ (buf2[0] & buf2[2]) ^ (buf2[1] & buf2[2]);
        unsigned int temp2 = f1 + maj;

        unsigned int f2 = ((buf2[4] >> 6)|(buf2[4] << 26)) ^ ((buf2[4] >> 11)|(buf2[4] << 21)) ^ ((buf2[4] >> 25)|(buf2[4] << 7));
        unsigned int ch = ((buf2[4] & buf2[5]) ^ (~buf2[4] & buf2[6]));
        unsigned int temp1 = buf2[7] + f2 + ch + sha_k[i] + schedules2[i];

//        unsigned int temp2 = SHA256_F1(buf[0]) + SHA2_MAJ(buf[0], buf[1], buf[2]);
//        unsigned int temp1 = buf[7] + SHA256_F2(buf[4]) + SHA2_CH(buf[4],buf[5],buf[6]) + sha256_k[i] + schedules2[i];
        buf2[7] = buf2[6];
        buf2[6] = buf2[5];
        buf2[5] = buf2[4];
        buf2[4] = buf2[3]+temp1;
        buf2[3] = buf2[2];
        buf2[2] = buf2[1];
        buf2[1] = buf2[0];
        buf2[0] = temp1+temp2;
    }

    for(i = 0; i < 8; i++) {
        sha256_h_buffer2[i] += buf2[i];
    }

    // ================================================================================================================================
    // ================================================================================================================================
    // comparing to the target

    unsigned int output[8];

    for (i = 0; i < 8; i++) {
        // reverse bytes needed for compare
        unsigned int b = sha256_h_buffer2[7-i];
        output[i] = (((b) >> 24)) + (((b << 8) >> 24) << 8) + (((b << 16) >> 24) << 16) + (((b << 24) >> 24) << 24);

        if(output[i] != 0 && target[i] != 0) {
            if (output[i] < target[i]) {
                // get header nonce
//                unsigned int headerNonce = 0;
//                for(int h = 0; h < 4; h++) {
//                    headerNonce += header[76+h] << (8*h);
//                }
                nonce[0] = testNonce;
                break;
            }
            break;
        }
        else if(output[i] != 0 && target[i] == 0) {
//            nonce[0] = 0;
            break;
        }
    }
}


__host__
int hexToDec(char hex) {
  switch(hex){
        case 'F'|'f':
        return 15;
        break;
        case 'E'|'e':
        return 14;
        break;
        case 'D'|'d':
        return 13;
        break;
        case 'C'|'c':
        return 12;
        break;
        case 'B'|'b':
        return 11;
        break;
        case 'A'|'a':
        return 10;
        break;
        case '9':
        return 9;
        break;
        case '8':
        return 8;
        break;
        case '7':
        return 7;
        break;
        case '6':
        return 6;
        break;
        case '5':
        return 5;
        break;
        case '4':
        return 4;
        break;
        case '3':
        return 3;
        break;
        case '2':
        return 2;
        break;
        case '1':
        return 1;
        break;
        default:
        return 0;
    }
}


int main(int argc, char* argv[])
{
  if (argc == 3) {

    unsigned int header_int[20];
    for (int i = 0; i < 20; i++) {
      int startPos = i*8;
      int byteOne = (hexToDec(argv[2][startPos]) * 16) + hexToDec(argv[2][startPos+1]);
      int byteTwo = (hexToDec(argv[2][startPos+2]) * 16) + hexToDec(argv[2][startPos+3]);
      int byteTre = (hexToDec(argv[2][startPos+4]) * 16) + hexToDec(argv[2][startPos+5]);
      int byteFor = (hexToDec(argv[2][startPos+6]) * 16) + hexToDec(argv[2][startPos+7]);
      header_int[i] = (byteOne << 24) + (byteTwo << 16) + (byteTre << 8) + byteFor;
    }

    unsigned int target_int[8];
    for (int i = 0; i < 8; i++) {
      int startPos = i*8;
      int byteOne = (hexToDec(argv[1][startPos]) * 16) + hexToDec(argv[1][startPos+1]);
      int byteTwo = (hexToDec(argv[1][startPos+2]) * 16) + hexToDec(argv[1][startPos+3]);
      int byteTre = (hexToDec(argv[1][startPos+4]) * 16) + hexToDec(argv[1][startPos+5]);
      int byteFor = (hexToDec(argv[1][startPos+6]) * 16) + hexToDec(argv[1][startPos+7]);
      target_int[i] = (byteOne << 24) + (byteTwo << 16) + (byteTre << 8) + byteFor;
    }


    // prepare memory for sha256 constants
    unsigned int *sha256_kp, *sha256_hp;
    hipMalloc(&sha256_kp, 64*sizeof(unsigned int));
    hipMemcpy(sha256_kp, &sha256_k, 64*sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMalloc(&sha256_hp, 8*sizeof(unsigned int));
    hipMemcpy(sha256_hp, &sha256_h, 8*sizeof(unsigned int), hipMemcpyHostToDevice);

    unsigned int *target, *header, *nonce;

    // prepare memory for header
    hipMalloc(&header, 20*sizeof(unsigned int));
    hipMemcpy(header, header_int, 20*sizeof(unsigned int), hipMemcpyHostToDevice);

    // prepare memory for target
    hipMalloc(&target, 8*sizeof(unsigned int));
    hipMemcpy(target, target_int, 8*sizeof(unsigned int), hipMemcpyHostToDevice);

    // prepare temporary nonce
    unsigned int *finalNonce;
    finalNonce = (unsigned int*)malloc(sizeof(unsigned int));
    hipMalloc(&nonce, sizeof(unsigned int));
    // cudaMemcpy(nonce, finalNonce, sizeof(unsigned int), cudaMemcpyHostToDevice);

    // N = elements
    // Perform SAXPY on 1M elements
    // saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);
    // int blocks = 64;
    // hash_header<<<(0xFFFFFFFF+1)/blocks, blocks>>>(sha256_kp, sha256_hp, target, header, nonce);

    // for (int i = 0; i < 8; i++) {
    //   std::cout << "target " << i << ": " << target_int[i] << std::endl;
    //   std::cout << "header " << i << ": " << header_int[i] << std::endl;
    // }

    const auto p1 = std::chrono::system_clock::now();

    int blocks = 128;
//    hash_header<<<(0xFFFFFFFF/blocks+2), blocks>>>(sha256_kp, sha256_hp, target, header, nonce);
    hash_header<<<1,1>>>(sha256_kp, sha256_hp, target, header, nonce);
    // hash_header<<<1000,1000>>>(sha256_kp, sha256_hp, target, header, nonce);
    hipMemcpy(finalNonce, nonce, sizeof(unsigned int), hipMemcpyDeviceToHost);


    const auto p2 = std::chrono::system_clock::now();
    int finaltime = (std::chrono::duration_cast<std::chrono::seconds>(p2.time_since_epoch()).count() - std::chrono::duration_cast<std::chrono::seconds>(p1.time_since_epoch()).count());

    std::cout << "time taken: " << finaltime << std::endl;

    // float maxError = 0.0f;
    // for (int i = 0; i < N; i++)
    //   maxError = max(maxError, abs(y[i]-4.0f));
    // printf("Max error: %f\n", maxError);

    std::cout << "0xFFFFFFFF       : " << 0xFFFFFFFF << std::endl;
    std::cout << "Final nonce found: " << finalNonce[0] << std::endl;

    hipFree(target);
    hipFree(header);
    hipFree(nonce);
    // free(target_int);
    // free(header_int);

  }
  else {
    return 0;
  }
  return 0;
}
